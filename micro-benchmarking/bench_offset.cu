/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

template <typename T>
__global__ void offset(T* a, int s)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x + s;
  a[i] = a[i]+1;
}

template <typename T>
__global__ void stride(T* a, int s)
{
  int i = (blockDim.x * blockIdx.x + threadIdx.x) * s;
  a[i] = a[i]+1;
}

template <typename T>
void runTest(int deviceId, int nMB)
{
  int blockSize = 256;
  float ms;

  T *d_a;
  hipEvent_t startEvent, stopEvent;
    
  int n = nMB*1024*1024/sizeof(T);

  // NB:  d_a(33*nMB) for stride case
  checkCuda( hipMalloc(&d_a, n * 33 * sizeof(T)) );

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  printf("Offset, Bandwidth (GB/s):\n");
  
  offset<<<n/blockSize, blockSize>>>(d_a, 0); // warm up
  offset<<<n/blockSize, blockSize>>>(d_a, 0);
  for (int i = 0; i <= 64; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    offset<<<n/blockSize, blockSize>>>(d_a, i%32);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  printf("\n");
  printf("Stride, Bandwidth (GB/s):\n");

  stride<<<n/blockSize, blockSize>>>(d_a, 1); // warm up
  for (int i = 1; i <= 32; i++) {
    checkCuda( hipMemset(d_a, 0, n * sizeof(T)) );

    checkCuda( hipEventRecord(startEvent,0) );
    stride<<<n/blockSize, blockSize>>>(d_a, i);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );

    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    printf("%d, %f\n", i, 2*nMB/ms);
  }

  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
  hipFree(d_a);
}

int main(int argc, char **argv)
{
  int nMB = 20;
  int deviceId = 0;
  bool bFp64 = false;

  for (int i = 1; i < argc; i++) {    
    if (!strncmp(argv[i], "dev=", 4))
      deviceId = atoi((char*)(&argv[i][4]));
    else if (!strcmp(argv[i], "fp64"))
      bFp64 = true;
  }
  
  hipDeviceProp_t prop;
  
  checkCuda( hipSetDevice(deviceId) );
  checkCuda( hipGetDeviceProperties(&prop, deviceId) );
  printf("Device: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", nMB);
  
  printf("%s Precision\n", bFp64 ? "Double" : "Single");
  
  if (bFp64) runTest<double>(deviceId, nMB);
  else       runTest<float>(deviceId, nMB);
}
